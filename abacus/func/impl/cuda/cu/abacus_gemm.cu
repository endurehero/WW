#include "hip/hip_runtime.h"
#include "abacus/func/impl/cuda/abacus_gemm.h"

namespace WW{
namespace abacus{

template<typename DataType>
__global__ void gemm(bool transA, bool transB, const int M, const int N, const int K, 
                    const float alpha, DataType* A, const float beta, const DataType* B,
                    float*C){
    

}


template<DATA_TYPE DataType>
ABACUS_STATUS AbacusGemm<NV, DataType>::forward(const std::vector<Tensor<NV>* >& inputs, std::vector<Tensor<NV>*>& outputs){

    LOG(INFO) << "Run Cuda Gemm Kernel.";
}


template class AbacusGemm<NV, DATA_FLOAT>;
} //namespace abacus
} //namespace WW